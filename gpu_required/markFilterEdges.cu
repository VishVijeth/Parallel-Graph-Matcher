
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void markFilterEdges_gpu(int * src, int * dst, int * matches, int * keepEdges, int numEdges) {
    
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
	int num_threads = blockDim.x * gridDim.x;
  
    for(int i = tid; i < numEdges; i+= num_threads){
       int j = src[i];
       int k = dst[i];
       if(matches[j] == -1 && matches[k] == -1){
           keepEdges[i] = 1;
       }
       else{
           keepEdges[i] = 0;
       }
   }
	
}
