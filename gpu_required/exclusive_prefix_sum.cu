
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void exclusive_prefix_sum_gpu(int * oldSum, int * newSum, int distance, int numElements) {
    
    int num_threads = blockDim.x * gridDim.x;
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    
    if(distance == 0){
        for(int i = tid; i < numElements ; i+= num_threads){
            if(i == 0){
                newSum[i] = 0;
            }
            else{
                newSum[i] = oldSum[i-1];
            } 
        }
    }
    else{
        for(int i = tid; i < numElements ; i+= num_threads){
                if(i-distance > 0){
                    newSum[i] = oldSum[i] + oldSum[i-distance];
                }
                else{
                    newSum[i] = oldSum[i];
                }
        }
    }
	
}
