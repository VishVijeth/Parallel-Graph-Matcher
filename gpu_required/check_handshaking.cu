
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void check_handshaking_gpu(int * strongNeighbor, int * matches, int numNodes) {
    /** YOUR CODE GOES BELOW **/
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int num_threads = blockDim.x * gridDim.x;

    for(int i = tid; i < numNodes; i+= num_threads){
        if(matches[i] == -1){
            int j = strongNeighbor[i];
            if((i == strongNeighbor[j])){
                matches[i] = j;
                matches[j] = i;
            }
        }
    }
	/** YOUR CODE GOES ABOVE **/
}
